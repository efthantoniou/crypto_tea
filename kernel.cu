#include "hip/hip_runtime.h"
#include "hiprand/hiprand.h"

#include "hiprand/hiprand_kernel.h"
#include "cuda_device_runtime_api.h"


#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <inttypes.h>
#include <assert.h>
#include <limits.h>
#include <math.h>

//Antoniou Efthimios 2022201300011 dit13011@uop.gr
//Makes the permutation that is applied to the key.

__device__ 
uint16_t key_permutation(uint16_t key) {
    uint16_t new_key = key & 0xF00F;
    new_key = new_key | ((key & 0x00F0) << 4) | ((key & 0x0F00) >> 4);
	return new_key;
}
//Makes a circular shift.
__device__ 
uint16_t circular_shift(uint16_t key) {
    uint16_t shift_key = 2;
    assert((shift_key < 16) && shift_key !=0);
	return  (key << shift_key) | (key >> (-shift_key & 15));
}
//Calculates the keys for all the rounds.
__device__ 
uint16_t *keys_enc_dec(uint16_t key){
    static uint16_t calc_keys[5]; 
    calc_keys[0] = key;
    calc_keys[1] = key_permutation(circular_shift(key));
    calc_keys[2] = key_permutation(circular_shift(calc_keys[1]));
    calc_keys[3] = key_permutation(circular_shift(calc_keys[2]));
    calc_keys[4] = circular_shift(calc_keys[3]);
    return calc_keys;
}
//Creates the s-box that permutates the bits of the ciphertext.
__device__  
uint16_t s_box(uint16_t ckey) {
    uint16_t new_key = ckey & 0x8421;
    new_key = new_key | ((ckey & 0x0842) << 3) | ((ckey & 0x1000) >> 9);
    new_key = new_key | ((ckey & 0x0084) << 6) | ((ckey & 0x2100) >> 6);
    new_key = new_key | ((ckey & 0x0008) << 9) | ((ckey & 0x4210) >> 3);
    return new_key;
}
//Encrypts the message that we provide.
__device__
uint16_t encrypt(uint16_t message, uint16_t *keys){
    //1st round
    uint16_t c_key = message ^ keys[0];
    c_key = s_box(c_key);
    //2nd round
    c_key = c_key ^ keys[1];
    c_key = s_box(c_key);
    //3rd round
    c_key = c_key ^ keys[2];
    c_key = s_box(c_key);
    //4th round
    c_key = c_key ^ keys[3];
    //5th round
    c_key = c_key ^ keys[4];
    return c_key;
}
//Decrypts the ciphertext that we provide.
__device__
void decrypt(uint16_t cipher, uint16_t *keys){
    uint16_t m_key;
    //1st round
    m_key = keys[4] ^ cipher;
    //2nd round
    m_key = m_key ^ keys[3];
    m_key = s_box(m_key);
    //3rd round
    m_key = m_key ^ keys[2];
    m_key = s_box(m_key);
    //4th round
    m_key = m_key ^ keys[1];
    m_key = s_box(m_key);
    //5th round
    m_key = m_key ^ keys[0];
    //return m_key; kanthn uint16_t an theleis na deis thn eksodo.

}

__global__ 
void kernel(float *devData){
	
	uint16_t cipher, keys_dev;
	size_t power26_dev= powf(2, 26);

	const int limit_16_dev = 65536;
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;

	int index = bid * blockDim.x + tid;
	int stride = blockDim.x * gridDim.x;

	keys_dev = 0xa1e9;

	for (int j = index; j < power26_dev;j+=stride) {
		cipher = encrypt((uint16_t)(devData[j]*limit_16_dev), keys_enc_dec(keys_dev));
		decrypt(cipher, keys_enc_dec(keys_dev));
	}
}

int main(void){
	hiprandGenerator_t gen;
	float *devData;
	size_t power26 = (size_t)pow(2, 26);
	//size_t free_byte, total_byte;

	/*cudaError_t cuda_status = cudaMemGetInfo(&free_byte, &total_byte);
	if (cudaSuccess != cuda_status) {
		printf("Error ");
		exit(1);
	}

	double used_db = (double)total_byte - (double)free_byte;
	printf("USED MEMORY: \t%f, FREE: \t%f, TOTAL: \t%f\n", used_db / 1024.0, (double)free_byte / 1024.0, (double)total_byte / 1024.0);
	*/
	printf("2");
	hipMalloc((void **)&devData, power26*sizeof(float));
	printf("1");
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, 6666667ULL);
	hiprandGenerateUniform(gen, devData, power26);

	kernel<<<128, 128 >>>(devData);

	hipDeviceSynchronize();
	/*
	cuda_status = cudaMemGetInfo(&free_byte, &total_byte);
	if (cudaSuccess != cuda_status) {
		printf("Error ");
		exit(1);
	}
	
	used_db = (double)total_byte-(double)free_byte;
	printf("USED MEMORY: \t%f, FREE: \t%f, TOTAL: \t%f\n",used_db/1024.0,(double)free_byte/1024.0,(double)total_byte/1024.0);
	*/
	hipFree(devData);

	hipDeviceReset();

	return 0;
}
